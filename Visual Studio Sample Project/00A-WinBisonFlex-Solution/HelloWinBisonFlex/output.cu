
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

__global__
void subtract(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] - y[i];
}

__global__
void multiply(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] * y[i];
}

__global__
void divide(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] / y[i];
}

void readData( char * filePath, int maxNumbers, float* buffer ){
	FILE * pFile;
	int i = 0;
	pFile = fopen(filePath, "r");

	while ( i < maxNumbers )
	{
		int number;
		if (fscanf(pFile, "%d", &number) != 1)
			break;        // file finished or there was an error
		buffer[i] = (float)number;
		i++;
	}
	fclose(pFile);
}

void writeData( char *filePath, int maxNumbers, float* buffer ){
	FILE * pFile;
	int i = 0;
	pFile = fopen(filePath, "w");

	while ( i < maxNumbers )
	{
		fprintf(pFile, "%d ", buffer[i]);
		i++;
	}
	fclose(pFile);
}

int main(void)
{
float * a;
hipMallocManaged(&a, 10000*sizeof(float));
float * b;
hipMallocManaged(&b, 10000*sizeof(float));
for ( int i = 0; i < 10000; i++ )
	b[i] = 10000.0f;
for ( int i = 0; i < 10000; i++ )
	a[i] = 5.0f;
readData( "valuesA.txt", 10000, a );
writeData( "valuesC.txt", 10000, a );
{ int blockSize = 256;
int numBlocks = ( 10000 + blockSize - 1) / blockSize;
add<<<numBlocks, blockSize>>>( 10000, a, b ); 
}
hipDeviceSynchronize();
for ( int i = 0; i < 10000; i++ )
	printf("%d ", a);
hipFree(a);
hipFree(b);
return 0;
}
